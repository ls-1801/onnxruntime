#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "quant_nbit_gemm.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

const int width_element_per_block = 32 * 2;
template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
  if (WarpSize >= 32)
    sum += __shfl_down_sync(0xffffffff, sum, 16);  // 0-16, 1-17, 2-18, etc.
  if (WarpSize >= 16)
    sum += __shfl_down_sync(0xffffffff, sum, 8);  // 0-8, 1-9, 2-10, etc.
  if (WarpSize >= 8)
    sum += __shfl_down_sync(0xffffffff, sum, 4);  // 0-4, 1-5, 2-6, etc.
  if (WarpSize >= 4)
    sum += __shfl_down_sync(0xffffffff, sum, 2);  // 0-2, 1-3, 4-6, 5-7, etc.
  if (WarpSize >= 2)
    sum += __shfl_down_sync(0xffffffff, sum, 1);  // 0-1, 2-3, 4-5, etc.
  return sum;
}
template <typename T>
__global__ void BatchGemv(T* out, const T* inA, const uint32_t* inB, const T* scales, const uint32_t* qzeros,
                          uint32_t groupsize, const uint32_t MATRIX_M, const uint32_t MATRIX_K, const uint32_t MATRIX_N) {
  const uint32_t block_k = ((MATRIX_K + 31) / 32 + 7) / 8 * 8;

  int bid = blockIdx.x;
  __shared__ float bsum[2][32][32 + 1];
  float sum[2] = {0, 0};
  int y_start = threadIdx.y * block_k;

  half2 res2 = {};
  half2 res2_1 = {};

  const half2* inA_start = (const half2*)(inA + blockIdx.y * MATRIX_K + y_start);

  int n_offset_x = bid * width_element_per_block + threadIdx.x * 2;

  int start_group_id = (y_start / groupsize);
  int compressed_idx = threadIdx.x % 4;
  half2 scale = ((const half2*)(scales + start_group_id * MATRIX_N + n_offset_x))[0];
  uint32_t qzero_p = ((qzeros + n_offset_x / 8 +
                       start_group_id * ((MATRIX_N + 7) / 8)))[0];
  half2 hzero = __halves2half2(
      __int2half_rn((qzero_p >> (8 * (compressed_idx))) & 0xf),
      __int2half_rn(((qzero_p) >> (8 * (compressed_idx) + 4)) & 0xf));
  half2 scale_h0 = __half2half2(scale.x);
  half2 scale_h1 = __half2half2(scale.y);
  half2 hzero_scale_0 = __half2half2(hzero.x * scale.x);
  half2 hzero_scale_1 = __half2half2(hzero.y * scale.y);

#pragma unroll
  for (int i = 0; i < block_k / 2; i += 4) {  // read half2 * 4
    res2 = {};
    res2_1 = {};
    int k_offset = y_start + i * 2;
    int g_id = k_offset / groupsize;

    if (g_id > start_group_id) {
      scale = ((const half2*)(scales + g_id * MATRIX_N + n_offset_x))[0];
      qzero_p = ((qzeros + n_offset_x / 8 +
                  g_id * ((MATRIX_N + 7) / 8)))[0];
      hzero = __halves2half2(
          __int2half_rn((qzero_p >> (8 * (compressed_idx))) & 0xf),
          __int2half_rn(((qzero_p) >> (8 * (compressed_idx) + 4)) & 0xf));
      scale_h0 = __half2half2(scale.x);
      scale_h1 = __half2half2(scale.y);
      hzero_scale_0 = __half2half2(hzero.x * scale.x);
      hzero_scale_1 = __half2half2(hzero.y * scale.y);
      start_group_id = g_id;
    }

    const uint32_t* hinB = inB + n_offset_x + k_offset / 8 * MATRIX_N;
    uint32_t vbInt1 =
        (n_offset_x < MATRIX_N && (k_offset < MATRIX_K)) ? hinB[0] : int32_t(0);
    uint32_t vbInt2 = (n_offset_x + 1 < MATRIX_N && (k_offset < MATRIX_K))
                          ? (hinB)[1]
                          : int32_t(0);
    half2 vb[8];
    const uint8_t* qweight_p1 = (const uint8_t*)&vbInt1;
    const uint8_t* qweight_p2 = (const uint8_t*)&vbInt2;

#pragma unroll
    for (int j = 0; j < 4; j++) {
      // vb[j] = __halves2half2(__int2half_rn(((vbInt1 >> (j * 8))) & 0xF),
      //                        __int2half_rn(((vbInt1) >> (j*8+4)) & 0xF));
      // vb[j + 4] = __halves2half2(__int2half_rn(((vbInt2)>>(j*8)) & 0xF),
      //                            __int2half_rn((((vbInt2) >> (j*8+4))) &
      //                            0xF));
      vb[j] = __halves2half2(__int2half_rn(((*(qweight_p1 + j))) & 0xF),
                             __int2half_rn(((*(qweight_p1 + j)) >> 4) & 0xF));
      vb[j + 4] =
          __halves2half2(__int2half_rn(((*(qweight_p2 + j))) & 0xF),
                         __int2half_rn((((*(qweight_p2 + j)) >> 4)) & 0xF));
    }

    half2 va[4];
    va[0] = (k_offset < MATRIX_K) ? ((inA_start))[i] : res2;
    va[1] = (k_offset + 1 < MATRIX_K) ? ((inA_start))[i + 1] : res2;
    va[2] = (k_offset + 2 < MATRIX_K) ? ((inA_start))[i + 2] : res2;
    va[3] = (k_offset + 3 < MATRIX_K) ? ((inA_start))[i + 3] : res2;

#pragma unroll
    for (int j = 0; j < 4; j++) {
      vb[j] = __hfma2(scale_h0, vb[j], -hzero_scale_0);
      res2 = __hfma2(va[j], vb[j], res2);
      vb[4 + j] = __hfma2(scale_h1, vb[4 + j], -hzero_scale_1);
      res2_1 = __hfma2(va[j], vb[4 + j], res2_1);
    }

    sum[0] += __half2float(res2.x) + __half2float(res2.y);
    sum[1] += __half2float(res2_1.x) + __half2float(res2_1.y);
  }
  // sum[0] += __half2float(res2.x);
  // sum[1] +=  __half2float(res2.y);
  bsum[0][threadIdx.x][threadIdx.y] = sum[0];
  bsum[1][threadIdx.x][threadIdx.y] = sum[1];

  __syncthreads();
  sum[0] = 0;
  sum[1] = 0;

#pragma unroll
  for (int i = 0; i < 2; i++) {
    sum[i] = bsum[i][threadIdx.y][threadIdx.x];
    __syncthreads();
    sum[i] = warpReduceSum<32>(sum[i]);
    if (threadIdx.x == 0) {
      out[+blockIdx.y * MATRIX_N + bid * width_element_per_block +
          threadIdx.y * 2 + i] = __float2half_rn(sum[i]);
    }
  }
}

void Q4bitGemv(
    hipStream_t stream,
    const void* vec_data,
    const int32_t* mat_data,
    void* mul_out_data,
    const void* scales_data,
    const int32_t* zeros_data,
    uint32_t MATRIX_M,
    uint32_t MATRIX_K,
    uint32_t MATRIX_N,
    uint32_t groupsize) {
  const int block_k = ((MATRIX_K + 31) / 32 + 7) / 8 * 8;

  dim3 gridDim = {(MATRIX_N + width_element_per_block - 1) / width_element_per_block, MATRIX_M};
  dim3 blockDim = {32, (MATRIX_K + block_k - 1) / block_k};
  BatchGemv<half><<<gridDim, blockDim, 0, stream>>>(
      static_cast<half*>(mul_out_data), static_cast<const half*>(vec_data),
      reinterpret_cast<const uint32_t*>(mat_data), static_cast<const half*>(scales_data),
      reinterpret_cast<const uint32_t*>(zeros_data), groupsize, MATRIX_M, MATRIX_K, MATRIX_N);
}

#if 0

#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ < 700
// adapted from https://github.com/torch/cutorch/blob/master/lib/THC/THCAtomics.cuh
//__device__ __forceinline__ void atomicAdd(__half* address, c10::Half val) {
//    unsigned int *address_as_ui = reinterpret_cast<unsigned int *>(reinterpret_cast<char *>(address) - (reinterpret_cast<size_t>(address) & 2));
//    unsigned int old = *address_as_ui;
//    unsigned int assumed;
//
//    do {
//        assumed = old;
//        unsigned short hsum = reinterpret_cast<size_t>(address) & 2 ? (old >> 16) : (old & 0xffff);
//        hsum += val;
//        old = reinterpret_cast<size_t>(address) & 2
//                 ? (old & 0xffff) | (hsum << 16)
//                 : (old & 0xffff0000) | hsum;
//        old = atomicCAS(address_as_ui, assumed, old);
//
//    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//    } while (assumed != old);
//}
#endif
#endif

const int BLOCKWIDTH = 256;
const int BLOCKHEIGHT4 = 32;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}

__global__ void VecQuant4MatMulKernel(
    const half2* __restrict__ vec,
    const int* __restrict__ mat,
    float* __restrict__ mul,
    const __half* __restrict__ scales,
    const int* __restrict__ zeros,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width,
    int groupsize) {
  const int blockwidth2 = BLOCKWIDTH / 2;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < blockwidth2)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[256][8];
  int val = threadIdx.x / 8;
  int off = threadIdx.x % 8;
  for (; val < 256; val += BLOCKWIDTH / 8) {
    deq2[val][off] = __halves2half2(__int2half_rn(val & 0xF), __int2half_rn(val >> 4));
  }

  int i = width * h + w;
  int g_h = h * 8;
  int k = 0;

  int z_w = w / 8;
  int z_mod = (w % 8) * 4;

  float res = 0;
  half2 res2;

  unsigned int tmp;

  __syncthreads();

  while (k < blockwidth2) {
    int g = (g_h + (k * 2)) / groupsize;
    float scale_f = (float)scales[g * width + w];
    half2 scale = __float2half2_rn(scale_f);
    half2 zero = __float2half2_rn(-(scale_f * (((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1)));

    res2 = {};
    tmp = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 0) & 0xff][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 8) & 0xff][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xff][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xff][off], scale, zero), blockvec[k + 3], res2);
    i += width;
    k += 4;
    res += __half2float(res2.x) + __half2float(res2.y);
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant4matmul_cuda(
    hipStream_t stream,
    const void* vec_data,
    const int* mat_data,
    void* mul_out_data,
    const void* scales_data,
    const int* zeros_data,
    int batch,
    int height,
    int width,
    int zero_width,
    int groupsize,
    int vec_height) {
  dim3 blocks(
      (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
      (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
      batch);
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernel<<<blocks, threads, 0, stream>>>(
      (const half2*)vec_data,
      mat_data,
      (float*)mul_out_data,
      (const __half*)scales_data,
      zeros_data,
      batch, vec_height, height, width, zero_width, groupsize);
}

__global__ void VecQuant4MatMulKernel_G(
    const half2* __restrict__ vec,
    const int* __restrict__ mat,
    float* __restrict__ mul,
    const __half* __restrict__ scales,
    const int* __restrict__ zeros,
    const int* __restrict__ g_idx,
    int batch,
    int vec_height,
    int height,
    int width,
    int zero_width) {
  const int blockwidth2 = BLOCKWIDTH / 2;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < blockwidth2)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[256][8];
  int val = threadIdx.x / 8;
  int off = threadIdx.x % 8;
  for (; val < 256; val += BLOCKWIDTH / 8) {
    deq2[val][off] = __halves2half2(
        __int2half_rn(val & 0xF), __int2half_rn(val >> 4));
  }

  int i = width * h + w;
  int g_h = h * 8;
  int k = 0;

  int z_w = w / 8;
  int z_mod = (w % 8) * 4;

  float res = 0;
  half2 res2;

  unsigned int tmp;

  __syncthreads();

  while (k < blockwidth2) {
    res2 = {};
    tmp = as_unsigned(mat[i]);

    int tmp_k = 0;
    half2 scales_tmp[4];
    half2 zeros_tmp[4];
    while (tmp_k < 4) {
      int g = as_int(g_idx[g_h + (k + tmp_k) * 2]);
      int g2 = as_int(g_idx[g_h + (k + tmp_k) * 2 + 1]);
      float scale_f = scales[g * width + w];
      float scale_f2 = scales[g2 * width + w];
      half2 scale = __halves2half2(scale_f, scale_f2);
      half2 zero = __halves2half2(
          __hmul(-scale_f, __int2half_rn(((as_unsigned(zeros[g * zero_width + z_w]) >> z_mod) & 0xF) + 1)),
          __hmul(-scale_f2, __int2half_rn(((as_unsigned(zeros[g2 * zero_width + z_w]) >> z_mod) & 0xF) + 1)));
      scales_tmp[tmp_k] = scale;
      zeros_tmp[tmp_k] = zero;
      tmp_k += 1;
    }

    res2 = __hfma2(__hfma2(deq2[(tmp >> 0) & 0xff][off], scales_tmp[0], zeros_tmp[0]), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 8) & 0xff][off], scales_tmp[1], zeros_tmp[1]), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xff][off], scales_tmp[2], zeros_tmp[2]), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xff][off], scales_tmp[3], zeros_tmp[3]), blockvec[k + 3], res2);
    i += width;
    k += 4;
    res = __hadd(res, (float)__hadd(res2.x, res2.y));
    ;
  }

  __half* mul2 = (__half*)mul;
  atomicAdd(&mul2[b * width + w], res);
}

void vecquant4matmul_g_cuda(
    hipStream_t stream,
    const void* vec_data,
    int* mat_data,
    const void* mul_out_data,
    const void* scales_data,
    int* zeros_data,
    int* g_idx_data,
    int batch,
    int height,
    int width,
    int zero_width,
    int vec_height) {
  dim3 blocks(
      (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
      (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
      batch);
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernel_G<<<blocks, threads, 0, stream>>>(
      (const half2*)vec_data,
      mat_data,
      (float*)mul_out_data,
      (const __half*)scales_data,
      zeros_data,
      g_idx_data,
      batch, vec_height, height, width, zero_width);
}
#endif

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
